// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.
#include "rocm_ops.hpp"
#include "mha_batch_decode.h"

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    MHA_BATCH_DECODE_PYBIND;
}