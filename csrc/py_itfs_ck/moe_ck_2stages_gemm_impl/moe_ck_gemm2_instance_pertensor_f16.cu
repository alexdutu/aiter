// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.
#include "moe_ck_gemm_common.cuh"

using A0DataType = F16;
using B0DataType = F16;
using AccDataType = F32;
using EDataType = F16;
using CDEElementOp = TypeCastExpertWeight;
const bool Nswizzle = false;
const bool PerTensorQuant = true;
CK_MOE_STAGE2_GEMM_DEFINE(32, 256/sizeof(A0DataType), 1, 4)
CK_MOE_STAGE2_GEMM_DEFINE(64, 256/sizeof(A0DataType), 1, 4)
CK_MOE_STAGE2_GEMM_DEFINE(128, 128/sizeof(A0DataType), 2, 2)


