// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include "py_itfs_common.h"
#include "moe_ck_gemm.hpp"

#define CK_MOE_STAGE1_GEMM_IMPL(A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, Nswizzle, isPerTensorQuant, ActOP, MPerBlock)                                                                                                                                                                                                                                    \
    if (isPerTensorQuant)                                                                                                                                                                                                                                                                                                                                               \
    {                                                                                                                                                                                                                                                                                                                                                                   \
        if (MPerBlock == 32)                                                                                                                                                                                                                                                                                                                                            \
            ck_moe_stage1_gemm<A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, 32, 256 / sizeof(A0DataType), 1, 4, Nswizzle, true, ActOP>(at::cuda::getCurrentCUDAStream().stream(), tokens, sorted_size, N, K, topk, hidden_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, num_valid_ids_ptr, out_ptr, w1_scale_ptr, a1_scale_ptr);   \
        else if (MPerBlock == 64)                                                                                                                                                                                                                                                                                                                                       \
            ck_moe_stage1_gemm<A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, 64, 256 / sizeof(A0DataType), 1, 4, Nswizzle, true, ActOP>(at::cuda::getCurrentCUDAStream().stream(), tokens, sorted_size, N, K, topk, hidden_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, num_valid_ids_ptr, out_ptr, w1_scale_ptr, a1_scale_ptr);   \
        else if (MPerBlock == 128)                                                                                                                                                                                                                                                                                                                                      \
            ck_moe_stage1_gemm<A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, 128, 128 / sizeof(A0DataType), 2, 2, Nswizzle, true, ActOP>(at::cuda::getCurrentCUDAStream().stream(), tokens, sorted_size, N, K, topk, hidden_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, num_valid_ids_ptr, out_ptr, w1_scale_ptr, a1_scale_ptr);  \
    }                                                                                                                                                                                                                                                                                                                                                                   \
    else                                                                                                                                                                                                                                                                                                                                                                \
    {                                                                                                                                                                                                                                                                                                                                                                   \
        if (MPerBlock == 32)                                                                                                                                                                                                                                                                                                                                            \
            ck_moe_stage1_gemm<A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, 32, 256 / sizeof(A0DataType), 1, 4, Nswizzle, false, ActOP>(at::cuda::getCurrentCUDAStream().stream(), tokens, sorted_size, N, K, topk, hidden_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, num_valid_ids_ptr, out_ptr, w1_scale_ptr, a1_scale_ptr);   \
        else if (MPerBlock == 64)                                                                                                                                                                                                                                                                                                                                       \
            ck_moe_stage1_gemm<A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, 64, 256 / sizeof(A0DataType), 1, 4, Nswizzle, false, ActOP>(at::cuda::getCurrentCUDAStream().stream(), tokens, sorted_size, N, K, topk, hidden_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, num_valid_ids_ptr, out_ptr, w1_scale_ptr, a1_scale_ptr);   \
        else if (MPerBlock == 128)                                                                                                                                                                                                                                                                                                                                      \
            ck_moe_stage1_gemm<A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, 128, 128 / sizeof(A0DataType), 2, 2, Nswizzle, false, ActOP>(at::cuda::getCurrentCUDAStream().stream(), tokens, sorted_size, N, K, topk, hidden_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, num_valid_ids_ptr, out_ptr, w1_scale_ptr, a1_scale_ptr);  \
    }

#define CK_MOE_STAGE1_GEMM_IMPL_INT4(A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, Nswizzle, isPerTensorQuant, ActOP, MPerBlock)                                                                                                                                                                                                                               \


void ck_moe_stage1(torch::Tensor &hidden_states,     // [m, k], input token
                   torch::Tensor &w1,                // [e, n, k]/[e, 2*n, k], pre-shuffle([e, nr, kr, w])
                   torch::Tensor &w2,                // [expert, dim, inter_dim], pre-shuffle([e, nr, kr, w])
                   torch::Tensor &sorted_token_ids,  // [max_num_tokens_padded]
                   torch::Tensor &sorted_expert_ids, // [max_num_m_blocks]
                   torch::Tensor &num_valid_ids,     // [1]
                   torch::Tensor &out,               // [m * topk, inter_dim]
                   int topk,
                   std::optional<torch::Tensor> w1_scale = std::nullopt, // [e, 1, n], gate(up) scale
                   std::optional<torch::Tensor> a1_scale = std::nullopt, // [m, 1], token scale
                   std::optional<int> block_m = 32)
{
    const at::cuda::OptionalCUDAGuard device_guard(device_of(out));
    at::cuda::getCurrentCUDAStream().stream();
    // TORCH_CHECK(hidden_states.dtype() == w1.dtype(),
    //             "Weights and activations should both be same dtype!");

    TORCH_CHECK(out.dtype() == at::ScalarType::BFloat16 || out.dtype() == at::ScalarType::Half,
                "Out dtype only support BFloat16/Float16!")

    int tokens = hidden_states.size(0);
    int sorted_size = sorted_token_ids.size(0);
    int E = w1.size(0);
    int N = w1.size(1) / 2;
    int K = hidden_states.size(-1);
    // int max_num_tokens_padded = sorted_token_ids.size(0);
    // int agvtokens_per_expert = max_num_tokens_padded / E;
    int MPerBlock = block_m.value();
    bool isPerTensorQuant = (!w1_scale.has_value()) || (w1_scale.value().numel() == E);

    // int M = agvtokens_per_expert < 32 ? 32 : (agvtokens_per_expert < 64 ? 64 : 128);

    void *hidden_states_ptr = hidden_states.data_ptr();
    void *w1_ptr = w1.transpose(1, 2).data_ptr();
    void *w2_ptr = w2.data_ptr();
    void *sorted_token_ids_ptr = sorted_token_ids.data_ptr();
    void *sorted_expert_ids_ptr = sorted_expert_ids.data_ptr();
    void *num_valid_ids_ptr = num_valid_ids.data_ptr();
    void *out_ptr = out.data_ptr();
    void *w1_scale_ptr = w1_scale.has_value() ? w1_scale.value().transpose(0, 1).data_ptr() : nullptr;
    void *a1_scale_ptr = a1_scale.has_value() ? a1_scale.value().data_ptr() : nullptr;

    // BF16
    if (hidden_states.dtype() == at::ScalarType::BFloat16)
    {
        using A0DataType = B16;
        using B0DataType = B16;
        using AccDataType = F32;
        using EDataType = B16;
        using CDEElementOp = TypeCast;
        const bool Nswizzle = false;
        const int ActOP = 2;
        CK_MOE_STAGE1_GEMM_IMPL(A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, Nswizzle, isPerTensorQuant, ActOP, MPerBlock);
    }
    // FP16
    else if (hidden_states.dtype() == at::ScalarType::Half)
    {
        using A0DataType = F16;
        using B0DataType = F16;
        using AccDataType = F32;
        using EDataType = F16;
        using CDEElementOp = TypeCast;
        const bool Nswizzle = false;
        const int ActOP = 2;
        CK_MOE_STAGE1_GEMM_IMPL(A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, Nswizzle, isPerTensorQuant, ActOP, MPerBlock);
    }
    // FP8 Wint4
    else if (hidden_states.dtype() == at::ScalarType::Float8_e4m3fnuz && w1.dtype() == at::ScalarType::UInt32)
    {
        using A0DataType = F8;
        using B0DataType = I4;
        const bool Nswizzle = false;
        TORCH_CHECK(a1_scale.has_value() && w1_scale.has_value(),
                    "MoE Quant must input scale!");
        TORCH_CHECK(a1_scale.value().dtype() == at::ScalarType::Float,
                    "Scales must be Float dtype!");
        using AccDataType = F32;
        using CDEElementOp = MulABScaleWint4;
        const int ActOP = 2;
        if (out.dtype() == at::ScalarType::Half)
        {
            CK_MOE_STAGE1_GEMM_IMPL_INT4(A0DataType, B0DataType, AccDataType, F16, CDEElementOp, Nswizzle, isPerTensorQuant, ActOP, MPerBlock);
        }
        else if (out.dtype() == at::ScalarType::BFloat16)
        {
            CK_MOE_STAGE1_GEMM_IMPL_INT4(A0DataType, B0DataType, AccDataType, B16, CDEElementOp, Nswizzle, isPerTensorQuant, ActOP, MPerBlock);
        }
    }
    // FP8
    else if (hidden_states.dtype() == at::ScalarType::Float8_e4m3fnuz)
    {
        using A0DataType = F8;
        using B0DataType = F8;
        TORCH_CHECK(a1_scale.has_value() && w1_scale.has_value(),
                    "MoE Quant must input scale!");
        TORCH_CHECK(a1_scale.value().dtype() == at::ScalarType::Float,
                    "Scales must be Float dtype!");
        using AccDataType = F32;
        using CDEElementOp = MulABScale;
        const bool Nswizzle = false;
        const int ActOP = 2;
        if (out.dtype() == at::ScalarType::Half)
        {
            CK_MOE_STAGE1_GEMM_IMPL(A0DataType, B0DataType, AccDataType, F16, CDEElementOp, Nswizzle, isPerTensorQuant, ActOP, MPerBlock);
        }
        else if (out.dtype() == at::ScalarType::BFloat16)
        {
            CK_MOE_STAGE1_GEMM_IMPL(A0DataType, B0DataType, AccDataType, B16, CDEElementOp, Nswizzle, isPerTensorQuant, ActOP, MPerBlock);
        }
    }
    // // I8
    // else if (hidden_states.dtype() == at::ScalarType::Char)
    // {
    //     using A0DataType = I8;
    //     using B0DataType = I8;
    //     TORCH_CHECK(a1_scale.has_value() && w1_scale.has_value(),
    //                 "MoE Quant must input scale!");
    //     TORCH_CHECK(a1_scale.value().dtype() == at::ScalarType::Float,
    //                 "Scales must be Float dtype!");
    //     using AccDataType = I32;
    //     using CDEElementOp = MulABScale;
    //     if (out.dtype() == at::ScalarType::Half)
    //     {
    //         CK_MOE_STAGE1_GEMM_IMPL(A0DataType, B0DataType, AccDataType, F16, CDEElementOp, MPerBlock);
    //     }
    //     else if (out.dtype() == at::ScalarType::BFloat16)
    //     {
    //         CK_MOE_STAGE1_GEMM_IMPL(A0DataType, B0DataType, AccDataType, B16, CDEElementOp, MPerBlock);
    //     }
    // }
}


#define CK_MOE_STAGE2_GEMM_IMPL(A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, Nswizzle, isPerTensorQuant, MPerBlock)                                                                                                                                                                                                                                                       \
    if (isPerTensorQuant)                                                                                                                                                                                                                                                                                                                                                                  \
    {                                                                                                                                                                                                                                                                                                                                                                                      \
        if (MPerBlock == 32)                                                                                                                                                                                                                                                                                                                                                               \
        {                                                                                                                                                                                                                                                                                                                                                               \
            if (K % (256 / sizeof(A0DataType)) == 0)                                                                                                                                                                                                                                                                                                                                                             \
            {                                                                                                                                                                                                                                                                                                                                                               \
                ck_moe_stage2_gemm<A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, 32, 256 / sizeof(A0DataType), 1, 4, Nswizzle, true>(at::cuda::getCurrentCUDAStream().stream(), tokens, sorted_size, N, K, topk, inter_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, sorted_weights_ptr, num_valid_ids_ptr, out_ptr, w2_scale_ptr, a2_scale_ptr);   \
            }                                                                                                                                                                                                                                                                                                                                                               \
        }                                                                                                                                                                                                                                                                                                                                                               \
        else if (MPerBlock == 64)                                                                                                                                                                                                                                                                                                                                                          \
        {                                                                                                                                                                                                                                                                                                                                                               \
            if (K % (256 / sizeof(A0DataType)) == 0)                                                                                                                                                                                                                                                                                                                                                             \
            {                                                                                                                                                                                                                                                                                                                                                               \
                ck_moe_stage2_gemm<A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, 64, 256 / sizeof(A0DataType), 1, 4, Nswizzle, true>(at::cuda::getCurrentCUDAStream().stream(), tokens, sorted_size, N, K, topk, inter_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, sorted_weights_ptr, num_valid_ids_ptr, out_ptr, w2_scale_ptr, a2_scale_ptr);   \
            }                                                                                                                                                                                                                                                                                                                                                               \
        }                                                                                                                                                                                                                                                                                                                                                               \
        else if (MPerBlock == 128)                                                                                                                                                                                                                                                                                                                                                         \
            ck_moe_stage2_gemm<A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, 128, 128 / sizeof(A0DataType), 2, 2, Nswizzle, true>(at::cuda::getCurrentCUDAStream().stream(), tokens, sorted_size, N, K, topk, inter_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, sorted_weights_ptr, num_valid_ids_ptr, out_ptr, w2_scale_ptr, a2_scale_ptr);  \
    }                                                                                                                                                                                                                                                                                                                                                                                      \
    else                                                                                                                                                                                                                                                                                                                                                                                   \
    {                                                                                                                                                                                                                                                                                                                                                                                      \
        if (MPerBlock == 32)                                                                                                                                                                                                                                                                                                                                                               \
        {                                                                                                                                                                                                                                                                                                                                                               \
            if (K % (256 / sizeof(A0DataType)) == 0)                                                                                                                                                                                                                                                                                                                                                             \
            {                                                                                                                                                                                                                                                                                                                                                               \
                ck_moe_stage2_gemm<A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, 32, 256 / sizeof(A0DataType), 1, 4, Nswizzle, false>(at::cuda::getCurrentCUDAStream().stream(), tokens, sorted_size, N, K, topk, inter_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, sorted_weights_ptr, num_valid_ids_ptr, out_ptr, w2_scale_ptr, a2_scale_ptr);   \
            }                                                                                                                                                                                                                                                                                                                                                               \
        }                                                                                                                                                                                                                                                                                                                                                               \
        else if (MPerBlock == 64)                                                                                                                                                                                                                                                                                                                                                          \
        {                                                                                                                                                                                                                                                                                                                                                               \
            if (K % (256 / sizeof(A0DataType)) == 0)                                                                                                                                                                                                                                                                                                                                                             \
            {                                                                                                                                                                                                                                                                                                                                                               \
                ck_moe_stage2_gemm<A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, 64, 256 / sizeof(A0DataType), 1, 4, Nswizzle, false>(at::cuda::getCurrentCUDAStream().stream(), tokens, sorted_size, N, K, topk, inter_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, sorted_weights_ptr, num_valid_ids_ptr, out_ptr, w2_scale_ptr, a2_scale_ptr);   \
            }                                                                                                                                                                                                                                                                                                                                                               \
        }                                                                                                                                                                                                                                                                                                                                                               \
        else if (MPerBlock == 128)                                                                                                                                                                                                                                                                                                                                                         \
            ck_moe_stage2_gemm<A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, 128, 128 / sizeof(A0DataType), 2, 2, Nswizzle, false>(at::cuda::getCurrentCUDAStream().stream(), tokens, sorted_size, N, K, topk, inter_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, sorted_weights_ptr, num_valid_ids_ptr, out_ptr, w2_scale_ptr, a2_scale_ptr); \
    }

#define CK_MOE_STAGE2_GEMM_IMPL_INT4(A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, Nswizzle, isPerTensorQuant, MPerBlock)                                                                                                                                                                                                                                                  \

void ck_moe_stage2(torch::Tensor &inter_states,      // [m, k], input token
                   torch::Tensor &w1,                // [e, n, k]/[e, 2*n, k], pre-shuffle([e, nr, kr, w])
                   torch::Tensor &w2,                // [expert, dim, inter_dim], pre-shuffle([e, nr, kr, w])
                   torch::Tensor &sorted_token_ids,  // [max_num_tokens_padded]
                   torch::Tensor &sorted_expert_ids, // [max_num_m_blocks]
                   torch::Tensor &sorted_weights,    // [max_num_tokens_padded]
                   torch::Tensor &num_valid_ids,     // [1]
                   torch::Tensor &out,               // [max_num_tokens_padded, inter_dim]
                   int topk,
                   std::optional<torch::Tensor> w2_scale = std::nullopt, // [e, 1, n], gate(up) scale
                   std::optional<torch::Tensor> a2_scale = std::nullopt, // [m, 1], token scale
                   std::optional<int> block_m = 32)
{
    // TORCH_CHECK(inter_states.dtype() == w2.dtype(),
    //             "Weights and activations should both be same dtype!");
    //
    TORCH_CHECK(out.dtype() == at::ScalarType::BFloat16 || out.dtype() == at::ScalarType::Half,
                "Out dtype only support BFloat16/Float16!")

    int tokens = inter_states.size(0);
    int sorted_size = sorted_token_ids.size(0);
    int E = w1.size(0);
    int N = w2.size(1);
    int K = inter_states.size(-1);
    // int max_num_tokens_padded = sorted_token_ids.size(0);
    // int agvtokens_per_expert = max_num_tokens_padded / E;
    int MPerBlock = block_m.value();
    // int M = agvtokens_per_expert < 32 ? 32 : (agvtokens_per_expert < 64 ? 64 : 128);
    bool isPerTensorQuant = (!w2_scale.has_value()) || (w2_scale.value().numel() == E);

    void *inter_states_ptr = inter_states.data_ptr();
    void *w1_ptr = w1.data_ptr();
    void *w2_ptr = w2.data_ptr();
    void *sorted_token_ids_ptr = sorted_token_ids.data_ptr();
    void *sorted_expert_ids_ptr = sorted_expert_ids.data_ptr();
    void *sorted_weights_ptr = sorted_weights.data_ptr();
    void *num_valid_ids_ptr = num_valid_ids.data_ptr();
    void *out_ptr = out.data_ptr();
    void *w2_scale_ptr = w2_scale.has_value() ? w2_scale.value().data_ptr() : nullptr;
    void *a2_scale_ptr = a2_scale.has_value() ? a2_scale.value().data_ptr() : nullptr;

    // BF16
    if (inter_states.dtype() == at::ScalarType::BFloat16)
    {
        using A0DataType = B16;
        using B0DataType = B16;
        using AccDataType = F32;
        using EDataType = B16;
        using CDEElementOp = TypeCastExpertWeight;
        const bool Nswizzle = false;
        CK_MOE_STAGE2_GEMM_IMPL(A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, Nswizzle, isPerTensorQuant, MPerBlock);
    }
    // FP16
    else if (inter_states.dtype() == at::ScalarType::Half)
    {
        using A0DataType = F16;
        using B0DataType = F16;
        using AccDataType = F32;
        using EDataType = F16;
        using CDEElementOp = TypeCastExpertWeight;
        const bool Nswizzle = false;
        CK_MOE_STAGE2_GEMM_IMPL(A0DataType, B0DataType, AccDataType, EDataType, CDEElementOp, Nswizzle, isPerTensorQuant, MPerBlock);
    }
    // FP8 wint4
    else if (inter_states.dtype() == at::ScalarType::Float8_e4m3fnuz && w1.dtype() == at::ScalarType::UInt32)
    {
        using A0DataType = F8;
        using B0DataType = I4;
        const bool Nswizzle = false;
        TORCH_CHECK(a2_scale.has_value() && w2_scale.has_value(),
                    "MoE Quant must input scale!");
        TORCH_CHECK(a2_scale.value().dtype() == at::ScalarType::Float,
                    "Scales must be Float dtype!");
        using AccDataType = F32;
        using CDEElementOp = MulABScaleExpertWeightWin4;
        if (out.dtype() == at::ScalarType::Half)
        {
            CK_MOE_STAGE2_GEMM_IMPL_INT4(A0DataType, B0DataType, AccDataType, F16, CDEElementOp, Nswizzle, isPerTensorQuant, MPerBlock);
        }
        else if (out.dtype() == at::ScalarType::BFloat16)
        {
            CK_MOE_STAGE2_GEMM_IMPL_INT4(A0DataType, B0DataType, AccDataType, B16, CDEElementOp, Nswizzle, isPerTensorQuant, MPerBlock);
        }
    }
    // FP8
    else if (inter_states.dtype() == at::ScalarType::Float8_e4m3fnuz)
    {
        using A0DataType = F8;
        using B0DataType = F8;
        TORCH_CHECK(a2_scale.has_value() && w2_scale.has_value(),
                    "MoE Quant must input scale!");
        TORCH_CHECK(a2_scale.value().dtype() == at::ScalarType::Float,
                    "Scales must be Float dtype!");
        using AccDataType = F32;
        using CDEElementOp = TypeCastExpertWeight;
        const bool Nswizzle = false;
        if (out.dtype() == at::ScalarType::Half)
        {
            CK_MOE_STAGE2_GEMM_IMPL(A0DataType, B0DataType, AccDataType, F16, CDEElementOp, Nswizzle, isPerTensorQuant, MPerBlock);
        }
        else if (out.dtype() == at::ScalarType::BFloat16)
        {
            CK_MOE_STAGE2_GEMM_IMPL(A0DataType, B0DataType, AccDataType, B16, CDEElementOp, Nswizzle, isPerTensorQuant, MPerBlock);
        }
    }
    // // I8
    // else if (inter_states.dtype() == at::ScalarType::Char)
    // {
    //     using A0DataType = I8;
    //     using B0DataType = I8;
    //     TORCH_CHECK(a2_scale.has_value() && w2_scale.has_value(),
    //                 "MoE Quant must input scale!");
    //     TORCH_CHECK(a2_scale.value().dtype() == at::ScalarType::Float,
    //                 "Scales must be Float dtype!");
    //     using AccDataType = I32;
    //     using CDEElementOp = MulABScaleExpertWeight;
    //     if (out.dtype() == at::ScalarType::Half)
    //     {
    //         CK_MOE_STAGE2_GEMM_IMPL(A0DataType, B0DataType, AccDataType, F16, CDEElementOp, MPerBlock);
    //     }
    //     else if (out.dtype() == at::ScalarType::BFloat16)
    //     {
    //         CK_MOE_STAGE2_GEMM_IMPL(A0DataType, B0DataType, AccDataType, B16, CDEElementOp, MPerBlock);
    //     }
    // }
}
